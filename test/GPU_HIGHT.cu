#include "hip/hip_runtime.h"
#include "HIGHT.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define _CRT_SECURE_NO_WARNINGS
#define HIGHT_BLOCK_SIZE    (8)

BYTE Delta[128] = {
       0x5A,0x6D,0x36,0x1B,0x0D,0x06,0x03,0x41,
       0x60,0x30,0x18,0x4C,0x66,0x33,0x59,0x2C,
       0x56,0x2B,0x15,0x4A,0x65,0x72,0x39,0x1C,
       0x4E,0x67,0x73,0x79,0x3C,0x5E,0x6F,0x37,
       0x5B,0x2D,0x16,0x0B,0x05,0x42,0x21,0x50,
       0x28,0x54,0x2A,0x55,0x6A,0x75,0x7A,0x7D,
       0x3E,0x5F,0x2F,0x17,0x4B,0x25,0x52,0x29,
       0x14,0x0A,0x45,0x62,0x31,0x58,0x6C,0x76,
       0x3B,0x1D,0x0E,0x47,0x63,0x71,0x78,0x7C,
       0x7E,0x7F,0x3F,0x1F,0x0F,0x07,0x43,0x61,
       0x70,0x38,0x5C,0x6E,0x77,0x7B,0x3D,0x1E,
       0x4F,0x27,0x53,0x69,0x34,0x1A,0x4D,0x26,
       0x13,0x49,0x24,0x12,0x09,0x04,0x02,0x01,
       0x40,0x20,0x10,0x08,0x44,0x22,0x11,0x48,
       0x64,0x32,0x19,0x0C,0x46,0x23,0x51,0x68,
       0x74,0x3A,0x5D,0x2E,0x57,0x6B,0x35,0x5A };

__device__ DWORD use_F0(DWORD X_before)
{
    DWORD use_XX1 = X_before;
    DWORD use_XX2 = X_before;
    DWORD use_XX3 = X_before;
    DWORD use_XX_F0;

    use_XX1 = (use_XX1 << 1) | (use_XX1 >> 7);
    use_XX2 = (use_XX2 << 2) | (use_XX2 >> 6);
    use_XX3 = (use_XX3 << 7) | (use_XX3 >> 1);
    use_XX_F0 = use_XX1 ^ use_XX2 ^ use_XX3;

    return use_XX_F0;
}

__device__ DWORD use_F1(DWORD X_before)
{
    DWORD use_XX4 = X_before;
    DWORD use_XX5 = X_before;
    DWORD use_XX6 = X_before;
    DWORD use_XX_F1;

    use_XX4 = (use_XX4 << 3) | (use_XX4 >> 5);
    use_XX5 = (use_XX5 << 4) | (use_XX5 >> 4);
    use_XX6 = (use_XX6 << 6) | (use_XX6 >> 2);
    use_XX_F1 = use_XX4 ^ use_XX5 ^ use_XX6;

    return use_XX_F1;
}

__device__ void    HIGHT_Encrypt(
    BYTE* roundkey,
    BYTE* pt,
    BYTE* ct)

{
    DWORD   XX[8];
    int us_id = 1;
    // First Round
    XX[1] = pt[1 * blockDim.x * gridDim.x];
    XX[3] = pt[3 * blockDim.x * gridDim.x];
    XX[5] = pt[5 * blockDim.x * gridDim.x];
    XX[7] = pt[7 * blockDim.x * gridDim.x];

    XX[0] = (pt[0 * blockDim.x * gridDim.x] + roundkey[0]) & 0xFF;
    XX[2] = (pt[2 * blockDim.x * gridDim.x] ^ roundkey[1]);
    XX[4] = (pt[4 * blockDim.x * gridDim.x] + roundkey[2]) & 0xFF;
    XX[6] = (pt[6 * blockDim.x * gridDim.x] ^ roundkey[3]);
    //printf("%02X, %02X, %02X, %02X, %02X, %02X, %02X, %02X\n", XX[0], XX[1], XX[2], XX[3], XX[4], XX[5], XX[6], XX[7]);
    // Encryption Round 
#define HIGHT_ENC(k, i0,i1,i2,i3,i4,i5,i6,i7) {                         \
        XX[i0] = (XX[i0] ^ (use_F0(XX[i1]) + roundkey[4*k+3])) & 0xFF;    \
        XX[i2] = (XX[i2] + (use_F1(XX[i3]) ^ roundkey[4*k+2])) & 0xFF;    \
        XX[i4] = (XX[i4] ^ (use_F0(XX[i5]) + roundkey[4*k+1])) & 0xFF;    \
        XX[i6] = (XX[i6] + (use_F1(XX[i7]) ^ roundkey[4*k+0])) & 0xFF;    \
    }

    HIGHT_ENC(2, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_ENC(3, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_ENC(4, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_ENC(5, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_ENC(6, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_ENC(7, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_ENC(8, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_ENC(9, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_ENC(10, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_ENC(11, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_ENC(12, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_ENC(13, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_ENC(14, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_ENC(15, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_ENC(16, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_ENC(17, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_ENC(18, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_ENC(19, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_ENC(20, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_ENC(21, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_ENC(22, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_ENC(23, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_ENC(24, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_ENC(25, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_ENC(26, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_ENC(27, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_ENC(28, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_ENC(29, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_ENC(30, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_ENC(31, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_ENC(32, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_ENC(33, 0, 7, 6, 5, 4, 3, 2, 1);

    // Final Round
    ct[1 * blockDim.x * gridDim.x] = (BYTE)XX[2];
    ct[3 * blockDim.x * gridDim.x] = (BYTE)XX[4];
    ct[5 * blockDim.x * gridDim.x] = (BYTE)XX[6];
    ct[7 * blockDim.x * gridDim.x] = (BYTE)XX[0];

    ct[0 * blockDim.x * gridDim.x] = (BYTE)(XX[1] + roundkey[4]);
    ct[2 * blockDim.x * gridDim.x] = (BYTE)(XX[3] ^ roundkey[5]);
    ct[4 * blockDim.x * gridDim.x] = (BYTE)(XX[5] + roundkey[6]);
    ct[6 * blockDim.x * gridDim.x] = (BYTE)(XX[7] ^ roundkey[7]);
}

__device__ void HIGHT_Decrypt(BYTE* RoundKey, BYTE* ct, BYTE* dt)
{
    DWORD   XX[8];



    XX[2] = (BYTE)ct[1 * blockDim.x * gridDim.x];
    XX[4] = (BYTE)ct[3 * blockDim.x * gridDim.x];
    XX[6] = (BYTE)ct[5 * blockDim.x * gridDim.x];
    XX[0] = (BYTE)ct[7 * blockDim.x * gridDim.x];

    XX[1] = (BYTE)(ct[0 * blockDim.x * gridDim.x] - RoundKey[4]);
    XX[3] = (BYTE)(ct[2 * blockDim.x * gridDim.x] ^ RoundKey[5]);
    XX[5] = (BYTE)(ct[4 * blockDim.x * gridDim.x] - RoundKey[6]);
    XX[7] = (BYTE)(ct[6 * blockDim.x * gridDim.x] ^ RoundKey[7]);
    //printf("%02X, %02X, %02X, %02X, %02X, %02X, %02X, %02X\n", XX[0], XX[1], XX[2], XX[3], XX[4], XX[5], XX[6], XX[7]);
#define HIGHT_DEC(k, i0,i1,i2,i3,i4,i5,i6,i7) {                         \
        XX[i1] = (XX[i1] - (use_F1(XX[i2]) ^ RoundKey[4*k+2])) & 0xFF;    \
        XX[i3] = (XX[i3] ^ (use_F0(XX[i4]) + RoundKey[4*k+1])) & 0xFF;    \
        XX[i5] = (XX[i5] - (use_F1(XX[i6]) ^ RoundKey[4*k+0])) & 0xFF;    \
        XX[i7] = (XX[i7] ^ (use_F0(XX[i0]) + RoundKey[4*k+3])) & 0xFF;    \
    }

    HIGHT_DEC(33, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_DEC(32, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_DEC(31, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_DEC(30, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_DEC(29, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_DEC(28, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_DEC(27, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_DEC(26, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_DEC(25, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_DEC(24, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_DEC(23, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_DEC(22, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_DEC(21, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_DEC(20, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_DEC(19, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_DEC(18, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_DEC(17, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_DEC(16, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_DEC(15, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_DEC(14, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_DEC(13, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_DEC(12, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_DEC(11, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_DEC(10, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_DEC(9, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_DEC(8, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_DEC(7, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_DEC(6, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_DEC(5, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_DEC(4, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_DEC(3, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_DEC(2, 6, 5, 4, 3, 2, 1, 0, 7);

    dt[1 * blockDim.x * gridDim.x] = (BYTE)(XX[1]);
    dt[3 * blockDim.x * gridDim.x] = (BYTE)(XX[3]);
    dt[5 * blockDim.x * gridDim.x] = (BYTE)(XX[5]);
    dt[7 * blockDim.x * gridDim.x] = (BYTE)(XX[7]);

    dt[0 * blockDim.x * gridDim.x] = (BYTE)(XX[0] - RoundKey[0]);
    dt[2 * blockDim.x * gridDim.x] = (BYTE)(XX[2] ^ RoundKey[1]);
    dt[4 * blockDim.x * gridDim.x] = (BYTE)(XX[4] - RoundKey[2]);
    dt[6 * blockDim.x * gridDim.x] = (BYTE)(XX[6] ^ RoundKey[3]);
}

__global__ void HIGHT_Encryption(unsigned char* key, unsigned char* pt, unsigned char* ct) 
{
    __shared__ BYTE GPU_rk[136];
    memcpy(GPU_rk, key, 136 * sizeof(BYTE));
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    HIGHT_Encrypt(GPU_rk, &pt[tid], &ct[tid]); 
}

__global__ void HIGHT_Decryption(unsigned char* key, unsigned char* ct, unsigned char* dt)
{
    __shared__ BYTE GPU_rk[136];
    memcpy(GPU_rk, key, 136 * sizeof(BYTE));
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    HIGHT_Decrypt(GPU_rk, ct + tid, dt + tid);
}


void    HIGHT_KeySched(
    BYTE* UserKey,
    DWORD   UserKeyLen,
    BYTE* RoundKey)
{
    int     i, j;

    for (i = 0; i < 4; i++) {
        RoundKey[i] = UserKey[i + 12];
        RoundKey[i + 4] = UserKey[i];
    }

    for (i = 0; i < 8; i++) {
        for (j = 0; j < 8; j++)
            RoundKey[8 + 16 * i + j] = (BYTE)(UserKey[(j - i) & 7] + Delta[16 * i + j]);
        // Use "&7"  instead of the "%8" for Performance

        for (j = 0; j < 8; j++)
            RoundKey[8 + 16 * i + j + 8] = (BYTE)(UserKey[((j - i) & 7) + 8] + Delta[16 * i + j + 8]);
    }
}

void HIGHT_GPU_performance_Test(unsigned long long Blocksize, unsigned long long Threadsize) {

    int i;

    hipEvent_t start, stop;
    float elapsed_time_ms = 0.0f;
    //CPU Memory
    unsigned char CPU_masterkey[16] = { 0x88, 0xE3, 0x4F, 0x8F, 0x08, 0x17, 0x79, 0xF1, 0xE9, 0xF3, 0x94, 0x37, 0x0A, 0xD4, 0x05, 0x89 };
    unsigned char CPU_roundkey[136] = { 0 };
    unsigned char* cpu_pt = NULL;
    cpu_pt = (unsigned char*)malloc(sizeof(unsigned char) * Blocksize * Threadsize * HIGHT_BLOCK_SIZE);

    unsigned char* us_CPU_pt = NULL;
    us_CPU_pt = (unsigned char*)malloc(sizeof(unsigned char) * Blocksize * Threadsize * HIGHT_BLOCK_SIZE);

    if (cpu_pt == NULL)
        return;
    for (int i = 0; i < Blocksize * Threadsize; i++) {
        cpu_pt[HIGHT_BLOCK_SIZE * i + 0] = 0xD7;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 1] = 0x6D;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 2] = 0x0D;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 3] = 0x18;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 4] = 0x32;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 5] = 0x7E;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 6] = 0xC5;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 7] = 0x62;
    }



    unsigned char* cpu_ct = NULL;
    cpu_ct = (unsigned char*)malloc(sizeof(unsigned char) * Blocksize * Threadsize * HIGHT_BLOCK_SIZE);
    unsigned char* cpu_dt = NULL;
    cpu_dt = (unsigned char*)malloc(sizeof(unsigned char) * Blocksize * Threadsize * HIGHT_BLOCK_SIZE);
    unsigned char* us_cpu_ct = NULL;
    us_cpu_ct = (unsigned char*)malloc(sizeof(unsigned char) * Blocksize * Threadsize * HIGHT_BLOCK_SIZE);
    unsigned char* us_cpu_dt = NULL;
    us_cpu_dt = (unsigned char*)malloc(sizeof(unsigned char) * Blocksize * Threadsize * HIGHT_BLOCK_SIZE);


    //GPU Memory
    //unsigned char* GPU_pt;
    unsigned char* GPU_ct;
    unsigned char* GPU_dt;
    unsigned char* GPU_roundkey;
    unsigned char* us_GPU_pt;

    //GPU memory allocation
    //hipMalloc((void**)&GPU_pt, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char));
    hipMalloc((void**)&GPU_ct, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char));
    hipMalloc((void**)&GPU_dt, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char));
    hipMalloc((void**)&GPU_roundkey, 136 * sizeof(unsigned char));

    hipMalloc((void**)&us_GPU_pt, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char));

    //CPU-> GPU Memory copy
    HIGHT_KeySched(CPU_masterkey, 16, CPU_roundkey); //roundkey�� �����

    hipMemcpy(GPU_roundkey, CPU_roundkey, 136 * sizeof(unsigned char), hipMemcpyHostToDevice);

    int k = 0;
    for (int i = 0; i < HIGHT_BLOCK_SIZE; i++)
    {
        for (int j = 0; j < Blocksize * Threadsize; j++)
        {
            us_CPU_pt[k++] = cpu_pt[HIGHT_BLOCK_SIZE * j + i];
        }
    }
    k = 0;

    hipMemcpy(us_GPU_pt, us_CPU_pt, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice);

    //Encryption
    printf("\n\nEncryption...\n");

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    HIGHT_Encryption <<<Blocksize, Threadsize >>> (GPU_roundkey, us_GPU_pt, GPU_ct);

    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    elapsed_time_ms /= 100;
    elapsed_time_ms = (Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char)) / elapsed_time_ms;
    elapsed_time_ms *= 1000;
    elapsed_time_ms /= (1024 * 1024 * 1024);
    printf("File size = %d MB, Grid : %d, Block : %d, Performance : %4.2f GB/s\n", (Blocksize * Threadsize * HIGHT_BLOCK_SIZE) / (1024 * 1024), Blocksize, Threadsize, elapsed_time_ms);
    getchar();
    getchar();

    hipMemcpy(cpu_ct, GPU_ct, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipGetLastError();
    hipDeviceSynchronize();

    for (int i = 0; i < Blocksize * Threadsize; i++)
    {
        us_cpu_ct[HIGHT_BLOCK_SIZE * i + 0] = cpu_ct[0 * Blocksize * Threadsize + i];
        us_cpu_ct[HIGHT_BLOCK_SIZE * i + 1] = cpu_ct[1 * Blocksize * Threadsize + i];
        us_cpu_ct[HIGHT_BLOCK_SIZE * i + 2] = cpu_ct[2 * Blocksize * Threadsize + i];
        us_cpu_ct[HIGHT_BLOCK_SIZE * i + 3] = cpu_ct[3 * Blocksize * Threadsize + i];
        us_cpu_ct[HIGHT_BLOCK_SIZE * i + 4] = cpu_ct[4 * Blocksize * Threadsize + i];
        us_cpu_ct[HIGHT_BLOCK_SIZE * i + 5] = cpu_ct[5 * Blocksize * Threadsize + i];
        us_cpu_ct[HIGHT_BLOCK_SIZE * i + 6] = cpu_ct[6 * Blocksize * Threadsize + i];
        us_cpu_ct[HIGHT_BLOCK_SIZE * i + 7] = cpu_ct[7 * Blocksize * Threadsize + i];
    }

    printf("HIGHT Cipher Text : \n");
    for (i = 0; i < Blocksize * Threadsize * HIGHT_BLOCK_SIZE; i++)
    {
        printf("%02X ", us_cpu_ct[i]);
        if ((i + 1) % 8 == 0)
        {
            printf("\n");
        }
    }

    getchar();
    printf("\n\nDecryption...\n");

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    HIGHT_Decryption << <Blocksize, Threadsize >> > (GPU_roundkey, GPU_ct, GPU_dt);

    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    elapsed_time_ms /= 100;
    elapsed_time_ms = (Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char)) / elapsed_time_ms;
    elapsed_time_ms *= 1000;
    elapsed_time_ms /= (1024 * 1024 * 1024);
    printf("File size = %d MB, Grid : %d, Block : %d, Performance : %4.2f GB/s\n", (Blocksize * Threadsize * HIGHT_BLOCK_SIZE) / (1024 * 1024), Blocksize, Threadsize, elapsed_time_ms);
    getchar();
    getchar();

    hipMemcpy(cpu_dt, GPU_dt, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char), hipMemcpyDeviceToHost);

    for (int i = 0; i < Blocksize * Threadsize; i++)
    {
        us_cpu_dt[HIGHT_BLOCK_SIZE * i + 0] = cpu_dt[0 * Blocksize * Threadsize + i];
        us_cpu_dt[HIGHT_BLOCK_SIZE * i + 1] = cpu_dt[1 * Blocksize * Threadsize + i];
        us_cpu_dt[HIGHT_BLOCK_SIZE * i + 2] = cpu_dt[2 * Blocksize * Threadsize + i];
        us_cpu_dt[HIGHT_BLOCK_SIZE * i + 3] = cpu_dt[3 * Blocksize * Threadsize + i];
        us_cpu_dt[HIGHT_BLOCK_SIZE * i + 4] = cpu_dt[4 * Blocksize * Threadsize + i];
        us_cpu_dt[HIGHT_BLOCK_SIZE * i + 5] = cpu_dt[5 * Blocksize * Threadsize + i];
        us_cpu_dt[HIGHT_BLOCK_SIZE * i + 6] = cpu_dt[6 * Blocksize * Threadsize + i];
        us_cpu_dt[HIGHT_BLOCK_SIZE * i + 7] = cpu_dt[7 * Blocksize * Threadsize + i];
    }

    printf("Plaintext  : \n");
    for (int i = 0; i < Blocksize * Threadsize * HIGHT_BLOCK_SIZE; i++) {
        printf("%02X ", us_cpu_dt[i]);
        if ((i + 1) % 8 == 0)
        {
            printf("\n");
        }
    }

    //hipFree(GPU_pt);
    hipFree(GPU_ct);
    hipFree(GPU_dt);
    hipFree(GPU_roundkey);
    hipFree(us_GPU_pt);
    free(cpu_ct);
    free(cpu_dt);
    free(cpu_pt);
    free(us_CPU_pt);
    free(us_cpu_ct);
    free(us_cpu_dt);
}

int main()
{
    unsigned long long BlockSize = 0, TreadSize = 0;
    printf("block�� ũ��: ");
    scanf("%d", &BlockSize);

    printf("tread�� ũ��: ");
    scanf("%d", &TreadSize);

    printf("\n");
    HIGHT_GPU_performance_Test(BlockSize, TreadSize);

    return 0;
}