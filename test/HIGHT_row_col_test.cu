#include "hip/hip_runtime.h"
#include "HIGHT.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define _CRT_SECURE_NO_WARNINGS
#define HIGHT_BLOCK_SIZE    (8)

BYTE Delta[128] = {
       0x5A,0x6D,0x36,0x1B,0x0D,0x06,0x03,0x41,
       0x60,0x30,0x18,0x4C,0x66,0x33,0x59,0x2C,
       0x56,0x2B,0x15,0x4A,0x65,0x72,0x39,0x1C,
       0x4E,0x67,0x73,0x79,0x3C,0x5E,0x6F,0x37,
       0x5B,0x2D,0x16,0x0B,0x05,0x42,0x21,0x50,
       0x28,0x54,0x2A,0x55,0x6A,0x75,0x7A,0x7D,
       0x3E,0x5F,0x2F,0x17,0x4B,0x25,0x52,0x29,
       0x14,0x0A,0x45,0x62,0x31,0x58,0x6C,0x76,
       0x3B,0x1D,0x0E,0x47,0x63,0x71,0x78,0x7C,
       0x7E,0x7F,0x3F,0x1F,0x0F,0x07,0x43,0x61,
       0x70,0x38,0x5C,0x6E,0x77,0x7B,0x3D,0x1E,
       0x4F,0x27,0x53,0x69,0x34,0x1A,0x4D,0x26,
       0x13,0x49,0x24,0x12,0x09,0x04,0x02,0x01,
       0x40,0x20,0x10,0x08,0x44,0x22,0x11,0x48,
       0x64,0x32,0x19,0x0C,0x46,0x23,0x51,0x68,
       0x74,0x3A,0x5D,0x2E,0x57,0x6B,0x35,0x5A };

// __device__ BYTE HIGHT_F0[256] = {
//        0x00,0x86,0x0D,0x8B,0x1A,0x9C,0x17,0x91,
//        0x34,0xB2,0x39,0xBF,0x2E,0xA8,0x23,0xA5,
//        0x68,0xEE,0x65,0xE3,0x72,0xF4,0x7F,0xF9,
//        0x5C,0xDA,0x51,0xD7,0x46,0xC0,0x4B,0xCD,
//        0xD0,0x56,0xDD,0x5B,0xCA,0x4C,0xC7,0x41,
//        0xE4,0x62,0xE9,0x6F,0xFE,0x78,0xF3,0x75,
//        0xB8,0x3E,0xB5,0x33,0xA2,0x24,0xAF,0x29,
//        0x8C,0x0A,0x81,0x07,0x96,0x10,0x9B,0x1D,
//        0xA1,0x27,0xAC,0x2A,0xBB,0x3D,0xB6,0x30,
//        0x95,0x13,0x98,0x1E,0x8F,0x09,0x82,0x04,
//        0xC9,0x4F,0xC4,0x42,0xD3,0x55,0xDE,0x58,
//        0xFD,0x7B,0xF0,0x76,0xE7,0x61,0xEA,0x6C,
//        0x71,0xF7,0x7C,0xFA,0x6B,0xED,0x66,0xE0,
//        0x45,0xC3,0x48,0xCE,0x5F,0xD9,0x52,0xD4,
//        0x19,0x9F,0x14,0x92,0x03,0x85,0x0E,0x88,
//        0x2D,0xAB,0x20,0xA6,0x37,0xB1,0x3A,0xBC,
//        0x43,0xC5,0x4E,0xC8,0x59,0xDF,0x54,0xD2,
//        0x77,0xF1,0x7A,0xFC,0x6D,0xEB,0x60,0xE6,
//        0x2B,0xAD,0x26,0xA0,0x31,0xB7,0x3C,0xBA,
//        0x1F,0x99,0x12,0x94,0x05,0x83,0x08,0x8E,
//        0x93,0x15,0x9E,0x18,0x89,0x0F,0x84,0x02,
//        0xA7,0x21,0xAA,0x2C,0xBD,0x3B,0xB0,0x36,
//        0xFB,0x7D,0xF6,0x70,0xE1,0x67,0xEC,0x6A,
//        0xCF,0x49,0xC2,0x44,0xD5,0x53,0xD8,0x5E,
//        0xE2,0x64,0xEF,0x69,0xF8,0x7E,0xF5,0x73,
//        0xD6,0x50,0xDB,0x5D,0xCC,0x4A,0xC1,0x47,
//        0x8A,0x0C,0x87,0x01,0x90,0x16,0x9D,0x1B,
//        0xBE,0x38,0xB3,0x35,0xA4,0x22,0xA9,0x2F,
//        0x32,0xB4,0x3F,0xB9,0x28,0xAE,0x25,0xA3,
//        0x06,0x80,0x0B,0x8D,0x1C,0x9A,0x11,0x97,
//        0x5A,0xDC,0x57,0xD1,0x40,0xC6,0x4D,0xCB,
//        0x6E,0xE8,0x63,0xE5,0x74,0xF2,0x79,0xFF };
//
//__device__ BYTE HIGHT_F1[256] = {
//        0x00,0x58,0xB0,0xE8,0x61,0x39,0xD1,0x89,
//        0xC2,0x9A,0x72,0x2A,0xA3,0xFB,0x13,0x4B,
//        0x85,0xDD,0x35,0x6D,0xE4,0xBC,0x54,0x0C,
//        0x47,0x1F,0xF7,0xAF,0x26,0x7E,0x96,0xCE,
//        0x0B,0x53,0xBB,0xE3,0x6A,0x32,0xDA,0x82,
//        0xC9,0x91,0x79,0x21,0xA8,0xF0,0x18,0x40,
//        0x8E,0xD6,0x3E,0x66,0xEF,0xB7,0x5F,0x07,
//        0x4C,0x14,0xFC,0xA4,0x2D,0x75,0x9D,0xC5,
//        0x16,0x4E,0xA6,0xFE,0x77,0x2F,0xC7,0x9F,
//        0xD4,0x8C,0x64,0x3C,0xB5,0xED,0x05,0x5D,
//        0x93,0xCB,0x23,0x7B,0xF2,0xAA,0x42,0x1A,
//        0x51,0x09,0xE1,0xB9,0x30,0x68,0x80,0xD8,
//        0x1D,0x45,0xAD,0xF5,0x7C,0x24,0xCC,0x94,
//        0xDF,0x87,0x6F,0x37,0xBE,0xE6,0x0E,0x56,
//        0x98,0xC0,0x28,0x70,0xF9,0xA1,0x49,0x11,
//        0x5A,0x02,0xEA,0xB2,0x3B,0x63,0x8B,0xD3,
//        0x2C,0x74,0x9C,0xC4,0x4D,0x15,0xFD,0xA5,
//        0xEE,0xB6,0x5E,0x06,0x8F,0xD7,0x3F,0x67,
//        0xA9,0xF1,0x19,0x41,0xC8,0x90,0x78,0x20,
//        0x6B,0x33,0xDB,0x83,0x0A,0x52,0xBA,0xE2,
//        0x27,0x7F,0x97,0xCF,0x46,0x1E,0xF6,0xAE,
//        0xE5,0xBD,0x55,0x0D,0x84,0xDC,0x34,0x6C,
//        0xA2,0xFA,0x12,0x4A,0xC3,0x9B,0x73,0x2B,
//        0x60,0x38,0xD0,0x88,0x01,0x59,0xB1,0xE9,
//        0x3A,0x62,0x8A,0xD2,0x5B,0x03,0xEB,0xB3,
//        0xF8,0xA0,0x48,0x10,0x99,0xC1,0x29,0x71,
//        0xBF,0xE7,0x0F,0x57,0xDE,0x86,0x6E,0x36,
//        0x7D,0x25,0xCD,0x95,0x1C,0x44,0xAC,0xF4,
//        0x31,0x69,0x81,0xD9,0x50,0x08,0xE0,0xB8,
//        0xF3,0xAB,0x43,0x1B,0x92,0xCA,0x22,0x7A,
//        0xB4,0xEC,0x04,0x5C,0xD5,0x8D,0x65,0x3D,
//        0x76,0x2E,0xC6,0x9E,0x17,0x4F,0xA7,0xFF };

__device__ DWORD use_F0(DWORD X_before)
{
    DWORD use_XX1 = X_before;
    DWORD use_XX2 = X_before;
    DWORD use_XX3 = X_before;
    DWORD use_XX_F0;

    use_XX1 = (use_XX1 << 1) | (use_XX1 >> 7);
    use_XX2 = (use_XX2 << 2) | (use_XX2 >> 6);
    use_XX3 = (use_XX3 << 7) | (use_XX3 >> 1);
    use_XX_F0 = use_XX1 ^ use_XX2 ^ use_XX3;

    return use_XX_F0;
}

__device__ DWORD use_F1(DWORD X_before)
{
    DWORD use_XX4 = X_before;
    DWORD use_XX5 = X_before;
    DWORD use_XX6 = X_before;
    DWORD use_XX_F1;

    use_XX4 = (use_XX4 << 3) | (use_XX4 >> 5);
    use_XX5 = (use_XX5 << 4) | (use_XX5 >> 4);
    use_XX6 = (use_XX6 << 6) | (use_XX6 >> 2);
    use_XX_F1 = use_XX4 ^ use_XX5 ^ use_XX6;

    return use_XX_F1;
}



__device__ void    HIGHT_Encrypt(
    BYTE* roundkey,
    BYTE* us_pt,
    BYTE* ct)

{
    DWORD   XX[8];
    BYTE pt[8];
    pt[0] = *(us_pt + 0);
    pt[1] = *(us_pt + 1);
    pt[2] = *(us_pt + 2);
    pt[3] = *(us_pt + 3);
    pt[4] = *(us_pt + 4);
    pt[5] = *(us_pt + 5);
    pt[6] = *(us_pt + 6);
    pt[7] = *(us_pt + 7);
    printf("%p\n", us_pt[0]);
    printf("%02X, %02X, %02X, %02X, %02X, %02X, %02X, %02X\n", pt[0], pt[1], pt[2], pt[3], pt[4], pt[5], pt[6], pt[7]);
    // First Round
    XX[1] = pt[1];
    XX[3] = pt[3];
    XX[5] = pt[5];
    XX[7] = pt[7];

    XX[0] = (pt[0] + roundkey[0]) & 0xFF;
    XX[2] = (pt[2] ^ roundkey[1]);
    XX[4] = (pt[4] + roundkey[2]) & 0xFF;
    XX[6] = (pt[6] ^ roundkey[3]);

    // Encryption Round 
#define HIGHT_ENC(k, i0,i1,i2,i3,i4,i5,i6,i7) {                         \
        XX[i0] = (XX[i0] ^ (use_F0(XX[i1]) + roundkey[4*k+3])) & 0xFF;    \
        XX[i2] = (XX[i2] + (use_F1(XX[i3]) ^ roundkey[4*k+2])) & 0xFF;    \
        XX[i4] = (XX[i4] ^ (use_F0(XX[i5]) + roundkey[4*k+1])) & 0xFF;    \
        XX[i6] = (XX[i6] + (use_F1(XX[i7]) ^ roundkey[4*k+0])) & 0xFF;    \
    }

    HIGHT_ENC(2, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_ENC(3, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_ENC(4, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_ENC(5, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_ENC(6, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_ENC(7, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_ENC(8, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_ENC(9, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_ENC(10, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_ENC(11, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_ENC(12, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_ENC(13, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_ENC(14, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_ENC(15, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_ENC(16, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_ENC(17, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_ENC(18, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_ENC(19, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_ENC(20, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_ENC(21, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_ENC(22, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_ENC(23, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_ENC(24, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_ENC(25, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_ENC(26, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_ENC(27, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_ENC(28, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_ENC(29, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_ENC(30, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_ENC(31, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_ENC(32, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_ENC(33, 0, 7, 6, 5, 4, 3, 2, 1);

    // Final Round
    ct[1] = (BYTE)XX[2];
    ct[3] = (BYTE)XX[4];
    ct[5] = (BYTE)XX[6];
    ct[7] = (BYTE)XX[0];

    ct[0] = (BYTE)(XX[1] + roundkey[4]);
    ct[2] = (BYTE)(XX[3] ^ roundkey[5]);
    ct[4] = (BYTE)(XX[5] + roundkey[6]);
    ct[6] = (BYTE)(XX[7] ^ roundkey[7]);
}

__device__ void HIGHT_Decrypt(BYTE* RoundKey, BYTE* ct, BYTE* dt)
{
    DWORD   XX[8];



    XX[2] = (BYTE)ct[1];
    XX[4] = (BYTE)ct[3];
    XX[6] = (BYTE)ct[5];
    XX[0] = (BYTE)ct[7];

    XX[1] = (BYTE)(ct[0] - RoundKey[4]);
    XX[3] = (BYTE)(ct[2] ^ RoundKey[5]);
    XX[5] = (BYTE)(ct[4] - RoundKey[6]);
    XX[7] = (BYTE)(ct[6] ^ RoundKey[7]);

#define HIGHT_DEC(k, i0,i1,i2,i3,i4,i5,i6,i7) {                         \
        XX[i1] = (XX[i1] - (use_F1(XX[i2]) ^ RoundKey[4*k+2])) & 0xFF;    \
        XX[i3] = (XX[i3] ^ (use_F0(XX[i4]) + RoundKey[4*k+1])) & 0xFF;    \
        XX[i5] = (XX[i5] - (use_F1(XX[i6]) ^ RoundKey[4*k+0])) & 0xFF;    \
        XX[i7] = (XX[i7] ^ (use_F0(XX[i0]) + RoundKey[4*k+3])) & 0xFF;    \
    }

    HIGHT_DEC(33, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_DEC(32, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_DEC(31, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_DEC(30, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_DEC(29, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_DEC(28, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_DEC(27, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_DEC(26, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_DEC(25, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_DEC(24, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_DEC(23, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_DEC(22, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_DEC(21, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_DEC(20, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_DEC(19, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_DEC(18, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_DEC(17, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_DEC(16, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_DEC(15, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_DEC(14, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_DEC(13, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_DEC(12, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_DEC(11, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_DEC(10, 6, 5, 4, 3, 2, 1, 0, 7);
    HIGHT_DEC(9, 7, 6, 5, 4, 3, 2, 1, 0);
    HIGHT_DEC(8, 0, 7, 6, 5, 4, 3, 2, 1);
    HIGHT_DEC(7, 1, 0, 7, 6, 5, 4, 3, 2);
    HIGHT_DEC(6, 2, 1, 0, 7, 6, 5, 4, 3);
    HIGHT_DEC(5, 3, 2, 1, 0, 7, 6, 5, 4);
    HIGHT_DEC(4, 4, 3, 2, 1, 0, 7, 6, 5);
    HIGHT_DEC(3, 5, 4, 3, 2, 1, 0, 7, 6);
    HIGHT_DEC(2, 6, 5, 4, 3, 2, 1, 0, 7);

    dt[1] = (BYTE)(XX[1]);
    dt[3] = (BYTE)(XX[3]);
    dt[5] = (BYTE)(XX[5]);
    dt[7] = (BYTE)(XX[7]);

    dt[0] = (BYTE)(XX[0] - RoundKey[0]);
    dt[2] = (BYTE)(XX[2] ^ RoundKey[1]);
    dt[4] = (BYTE)(XX[4] - RoundKey[2]);
    dt[6] = (BYTE)(XX[6] ^ RoundKey[3]);
}

__global__ void HIGHT_Encryption(unsigned char* key, unsigned char* pt, unsigned char* ct) {
    __shared__ BYTE GPU_rk[136];
    memcpy(GPU_rk, key, 136 * sizeof(BYTE));
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //HIGHT_Encrypt(GPU_rk, pt + tid, ct + tid);
    HIGHT_Encrypt(GPU_rk, pt + tid, ct + tid);

}

__global__ void HIGHT_Decryption(unsigned char* key, unsigned char* ct, unsigned char* dt)
{
    __shared__ BYTE GPU_rk[136];
    memcpy(GPU_rk, key, 136 * sizeof(BYTE));
    int tid = threadIdx.x * HIGHT_BLOCK_SIZE + blockIdx.x * blockDim.x * HIGHT_BLOCK_SIZE;
    HIGHT_Decrypt(GPU_rk, ct + tid, dt + tid);
}

void    HIGHT_KeySched(
    BYTE* UserKey,
    DWORD   UserKeyLen,
    BYTE* RoundKey)
{
    int     i, j;

    for (i = 0; i < 4; i++) {
        RoundKey[i] = UserKey[i + 12];
        RoundKey[i + 4] = UserKey[i];
    }

    for (i = 0; i < 8; i++) {
        for (j = 0; j < 8; j++)
            RoundKey[8 + 16 * i + j] = (BYTE)(UserKey[(j - i) & 7] + Delta[16 * i + j]);

        for (j = 0; j < 8; j++)
            RoundKey[8 + 16 * i + j + 8] = (BYTE)(UserKey[((j - i) & 7) + 8] + Delta[16 * i + j + 8]);
    }
}

void HIGHT_GPU_performance_Test(unsigned long long Blocksize, unsigned long long Threadsize) {

    int i;

    hipEvent_t start, stop;
    float elapsed_time_ms = 0.0f;
    //CPU Memory
    unsigned char CPU_masterkey[16] = { 0x88, 0xE3, 0x4F, 0x8F, 0x08, 0x17, 0x79, 0xF1, 0xE9, 0xF3, 0x94, 0x37, 0x0A, 0xD4, 0x05, 0x89 };
    unsigned char CPU_roundkey[136] = { 0 };
    unsigned char* cpu_pt = NULL;
    cpu_pt = (unsigned char*)malloc(sizeof(unsigned char) * Blocksize * Threadsize * HIGHT_BLOCK_SIZE);
    if (cpu_pt == NULL)
        return;
    for (int i = 0; i < Blocksize * Threadsize; i++) {
        cpu_pt[HIGHT_BLOCK_SIZE * i + 0] = 0xD7;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 1] = 0x6D;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 2] = 0x0D;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 3] = 0x18;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 4] = 0x32;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 5] = 0x7E;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 6] = 0xC5;
        cpu_pt[HIGHT_BLOCK_SIZE * i + 7] = 0x62;
    }
    unsigned char* cpu_ct = NULL;
    cpu_ct = (unsigned char*)malloc(sizeof(unsigned char) * Blocksize * Threadsize * HIGHT_BLOCK_SIZE);
    unsigned char* cpu_dt = NULL;
    cpu_dt = (unsigned char*)malloc(sizeof(unsigned char) * Blocksize * Threadsize * HIGHT_BLOCK_SIZE);


    unsigned char* us_cpu_pt = NULL;
    us_cpu_pt = (unsigned char*)malloc(sizeof(unsigned char) * Blocksize * Threadsize * HIGHT_BLOCK_SIZE);

    //GPU Memory
    unsigned char* GPU_pt;
    unsigned char* GPU_ct;
    unsigned char* GPU_dt;
    unsigned char* GPU_roundkey;

    unsigned char* us_GPU_pt;

    //GPU memory allocation
    hipMalloc((void**)&GPU_pt, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char));
    hipMalloc((void**)&GPU_ct, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char));
    hipMalloc((void**)&GPU_dt, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char));
    hipMalloc((void**)&GPU_roundkey, 136 * sizeof(unsigned char));

    hipMalloc((void**)&us_GPU_pt, Blocksize * Threadsize * sizeof(unsigned char));

    //CPU-> GPU Memory copy
    HIGHT_KeySched(CPU_masterkey, 16, CPU_roundkey); //roundkey�� �����

    hipMemcpy(GPU_pt, cpu_pt, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(GPU_roundkey, CPU_roundkey, 136 * sizeof(unsigned char), hipMemcpyHostToDevice);

    printf("\n\nEncryption...\n");

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    HIGHT_Encryption << <Blocksize, Threadsize >> > (GPU_roundkey, GPU_pt, GPU_ct);

    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    elapsed_time_ms /= 100;
    elapsed_time_ms = (Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char)) / elapsed_time_ms;
    elapsed_time_ms *= 1000;
    elapsed_time_ms /= (1024 * 1024 * 1024);
    printf("File size = %d MB, Grid : %d, Block : %d, Performance : %4.2f GB/s\n", (Blocksize * Threadsize * HIGHT_BLOCK_SIZE) / (1024 * 1024), Blocksize, Threadsize, elapsed_time_ms);
    getchar();
    getchar();

    hipMemcpy(cpu_ct, GPU_ct, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipGetLastError();
    hipDeviceSynchronize();

    printf("HIGHT Cipher Text : \n");
    for (i = 0; i < Blocksize * Threadsize * HIGHT_BLOCK_SIZE; i++)
    {
        printf("%02X ", cpu_ct[i]);
        if ((i + 1) % 8 == 0)
        {
            printf("\n");
        }
    }
    printf("\n");

    getchar();
    printf("\n\nDecryption...\n");

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    HIGHT_Decryption << <Blocksize, Threadsize >> > (GPU_roundkey, GPU_ct, GPU_dt);

    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    elapsed_time_ms /= 100;
    elapsed_time_ms = (Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char)) / elapsed_time_ms;
    elapsed_time_ms *= 1000;
    elapsed_time_ms /= (1024 * 1024 * 1024);
    printf("File size = %d MB, Grid : %d, Block : %d, Performance : %4.2f GB/s\n", (Blocksize * Threadsize * HIGHT_BLOCK_SIZE) / (1024 * 1024), Blocksize, Threadsize, elapsed_time_ms);
    getchar();
    getchar();

    hipMemcpy(cpu_dt, GPU_dt, Blocksize * Threadsize * HIGHT_BLOCK_SIZE * sizeof(unsigned char), hipMemcpyDeviceToHost);
    printf("Plaintext  : \n");
    for (int i = 0; i < Blocksize * Threadsize * HIGHT_BLOCK_SIZE; i++) {
        printf("%02X ", cpu_dt[i]);
        if ((i + 1) % 8 == 0)
        {
            printf("\n");
        }
    }

    hipFree(GPU_pt);
    hipFree(GPU_ct);
    hipFree(GPU_dt);
    hipFree(GPU_roundkey);
    hipFree(us_GPU_pt);
    free(cpu_ct);
    free(cpu_dt);
    free(cpu_pt);
    free(us_cpu_pt);
}

int main()
{
    unsigned long long BlockSize = 0, TreadSize = 0;
    printf("block�� ũ��: ");
    scanf("%d", &BlockSize);

    printf("tread�� ũ��: ");
    scanf("%d", &TreadSize);

    printf("\n");
    HIGHT_GPU_performance_Test(BlockSize, TreadSize);

    return 0;
}